#include "hip/hip_runtime.h"
// 实现一个矩阵乘法

__global__ void matmul(float* A, float* B, float*C, int M, int N, int K) {
    // 计算行列号
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockDim.y * blockIdx.y + threadIdx.y;

    // 边界条件,避免越界
    if (row >= M || col >= N) {
        return;
    }

    // A的row行k列 B的k行col列
    float acc = 0;
    for (int k = 0; k < K; ++k) {
        acc = A[row * K + k] * B[k * N + col];
    }

    // C的row行col列
    C[row * N + col] = out;
}

// 向上取整的除法
#define CEIL_DIV(a, b) ((a + b - 1) / b)

int main() {
    // 初始化矩阵A，B，C

    dim3 blockDim(16, 16);
    dim3 gridDim(CEIL_DIV(blockDim.x), CEIL_DIV(blockDim.y));
    <<<gridDim, blockDim>>>matmul(A, B, C, M, N, K);

    return 0;
}

